#include "hip/hip_runtime.h"
#include "fast_int_math_v2.hpp"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>
#include <vector>

constexpr size_t TEST_SQRT_STEP = 1 << 26;
constexpr size_t TEST_DIV_STEP = 1 << 21;
constexpr int NUMBERS_PER_DIVISOR = 256;

// Run "generate_ptx.bat" and search for "fast_div_v2 BEGIN" in kernel.ptx to look at generated PTX assembly
__global__ void DummyFastDivPTX(const uint64_t* _a, const uint32_t* _b, uint64_t* _result)
{
	__shared__ uint32_t RCP[256];
	for (int i = threadIdx.x; i < 256; i += blockDim.x)
	{
		RCP[i] = RCP_C[i];
	}
	__syncthreads();

	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	const uint64_t a = _a[index];
	const uint32_t b = _b[index];
	uint64_t result;

	asm("// fast_div_v2 BEGIN");
	result = fast_div_v2(RCP, a, b);
	asm("// fast_div_v2 END");

	_result[index] = result;
}

// Run "generate_ptx.bat" and search for "fast_sqrt_v2 BEGIN" in kernel.ptx to look at generated PTX assembly
__global__ void DummyFastSqrtPTX(const uint64_t* _a, uint32_t* _result)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	const uint64_t a = _a[index];
	uint32_t result;

	asm("// fast_sqrt_v2 BEGIN");
	result = fast_sqrt_v2(a);
	asm("// fast_sqrt_v2 END");

	_result[index] = result;
}

__global__ void FastDivTest(const uint64_t* _a, const uint32_t base, uint64_t* err_value)
{
	__shared__ uint32_t RCP[256];
	for (int i = threadIdx.x; i < 256; i += blockDim.x)
	{
		RCP[i] = RCP_C[i];
	}
	__syncthreads();

	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	const uint64_t a = _a[index % NUMBERS_PER_DIVISOR];
	uint32_t b = base + (index / NUMBERS_PER_DIVISOR);
	if (b == 0x80000000UL) b = 0x80000001UL;

	const uint64_t fast_div_result = fast_div_v2(RCP, a, b);
	const uint64_t correct_result = (uint64_t(a % b) << 32) + (uint32_t)(a / b);
	if ((fast_div_result != correct_result) && (atomicAdd((uint32_t*)err_value, 1) == 0))
	{
		err_value[1] = a;
		err_value[2] = b;
		err_value[3] = fast_div_result;
		err_value[4] = correct_result;
	}
}

__device__ __forceinline__ void report_sqrt_error(uint64_t i, uint64_t n, uint64_t expected, uint64_t actual, uint64_t* err_value)
{
	if (atomicAdd((uint32_t*)(err_value), 1) == 0)
	{
		err_value[1] = i;
		err_value[2] = n;
		err_value[3] = expected;
		err_value[4] = actual;
	}
}

__global__ void FastSqrtTest(const uint32_t base, uint64_t* err_value)
{
	const int i = base + blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= 1779033703)
	{
		if (i == 1779033703)
		{
			const uint64_t n1 = (uint64_t)(-1);
			const uint32_t r1 = fast_sqrt_v2(n1);
			if (r1 != 3558067407U) { report_sqrt_error(i, n1, 3558067407, r1, err_value); }
		}
		return;
	}

	const uint64_t i1 = i + (1ULL << 32);
	const uint64_t n1 = i1 * i1;

	const uint32_t r1 = fast_sqrt_v2(n1);
	if (r1 != i1 * 2 - (1ULL << 33)) { report_sqrt_error(i, n1, i1 * 2 - (1ULL << 33), r1, err_value); }

	const uint32_t r2 = fast_sqrt_v2(n1 + i1);
	if (r2 != i1 * 2 - (1ULL << 33)) { report_sqrt_error(i, n1 + i1, i1 * 2 - (1ULL << 33), r2, err_value); }

	const uint32_t r3 = fast_sqrt_v2(n1 + i1 + 1);
	if (r3 != i1 * 2 + 1 - (1ULL << 33)) { report_sqrt_error(i, n1 + i1 + 1, i1 * 2 + 1 - (1ULL << 33), r2, err_value); }

	const uint64_t i2 = i + (1ULL << 32) + 1;
	const uint64_t n2 = i2 * i2 - 1;

	const uint32_t r4 = fast_sqrt_v2(n2);
	if (r4 != i2 * 2 - 1 - (1ULL << 33)) { report_sqrt_error(i, n2, i2 * 2 - 1 - (1ULL << 33), r4, err_value); }
}

hipError_t TestIntMath()
{
	uint64_t a[NUMBERS_PER_DIVISOR];
	a[0] = 0;
	a[1] = uint64_t(-1);
	uint64_t k = 11400714819323198485ULL;
	for (int i = 2; i < NUMBERS_PER_DIVISOR; ++i)
	{
		a[i] = k;
		k = k * 2862933555777941757ULL + 3037000493;
	}

	uint64_t* dev_a = nullptr;
	uint64_t* dev_err_value = nullptr;

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, sizeof(uint64_t) * NUMBERS_PER_DIVISOR);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_err_value, sizeof(uint64_t) * 5);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, sizeof(uint64_t) * NUMBERS_PER_DIVISOR, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	printf("Testing fast_sqrt_v2 (all edge cases)\n");
	for (uint32_t base = 0; base <= 1779033703; base += TEST_SQRT_STEP)
	{
		printf("%.1f%% done\r", base * 100.0 / 1779033703);

		FastSqrtTest<<<TEST_SQRT_STEP / 256, 256>>>(base, dev_err_value);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "FastSqrtTest launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		uint64_t err_value[5] = {};
		cudaStatus = hipMemcpy(err_value, dev_err_value, sizeof(uint64_t) * 5, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		if (err_value[0] != 0)
		{
			printf("\nFailed for i=%llu, N=%llu: expected %llu, got %llu\n", err_value[1], err_value[2], err_value[3], err_value[4]);
			goto Error;
		}
	}

	printf("Testing fast_div_v2 (all divisors, %d numbers per divisor)\n", NUMBERS_PER_DIVISOR);
	for (uint64_t base = 0x80000000ULL; base < 0x100000000ULL; base += TEST_DIV_STEP)
	{
		printf("%.1f%% done\r", (base - 0x80000000ULL) * 100.0 / (0x100000000ULL - 0x80000000ULL));

		FastDivTest<<<(TEST_DIV_STEP * NUMBERS_PER_DIVISOR) / 256, 256>>>(dev_a, (uint32_t)(base), dev_err_value);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "FastDivTest launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		uint64_t err_value[5] = {};
		cudaStatus = hipMemcpy(err_value, dev_err_value, sizeof(uint64_t) * 5, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		if (err_value[0] != 0)
		{
			printf("\nError:\na=%llu\nb=%llu\nfast_div result=%llu\ncorrect result=%llu\n", err_value[1], err_value[2], err_value[3], err_value[4]);
			goto Error;
		}
	}
	printf("100.0%% done\n");

Error:
	hipFree(dev_a);
	hipFree(dev_err_value);

	return cudaStatus;
}

int main()
{
    hipError_t cudaStatus = TestIntMath();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "TestIntMath failed!");
        return 1;
    }

	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
